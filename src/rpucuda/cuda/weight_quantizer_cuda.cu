#include "hip/hip_runtime.h"
/**
 * 
 * @file weight_quantizer_cuda.cu
 * @brief Weight quantizer CUDA kernel implementations.
 * 
 */


#include "cuda_fp16_util.h"
#include "cuda_math_util.h"
#include "io_iterator.h"
#include "rpu_cub.h"
#include "weight_quantizer_cuda.h"


namespace RPU {

template <typename T>
WeightQuantizerCuda<T>::WeightQuantizerCuda(CudaContextPtr context, int x_size, int d_size)
    : context_(context), x_size_(x_size), d_size_(d_size), size_(x_size * d_size) {

//   T *tmp = nullptr;
//   StdFunctor<T> std_functor((T)x_size_, tmp);
//   RPU_CUB_NS_QUALIFIER TransformInputIterator<T, StdFunctor<T>, T *> std_input(tmp, std_functor);

//   RPU_CUB_NS_QUALIFIER DeviceReduce::Sum(
//       nullptr, temp_storage_bytes_, std_input, tmp, size_, context_->getStream());
//   dev_temp_storage_ = RPU::make_unique<CudaArray<char>>(context, temp_storage_bytes_);
}

template <typename T>
__global__ T WeightQuantizerCuda<T>::fit(const T *weights, const WeightQuantizerParameter<T> &wqpar, const T bound) {

    // The fit function is used to fine tune the redolution of the quantizer, so that up to a minimum
    // of (1 - eps) fraction of the weights are included in the FSR.

    if (wqpar.resolution != 0 || wqpar.eps == 0){
        return wqpar.resolution;
    }

    int total_weights = size_;
    T percentage = (float)wqpar.eps;
    int max_count = (int)(total_weights * percentage/2.);

    std::vector<T> sorted_weights(size_);
    PRAGMA_SIMD
    for (int i = 0; i < size_; i++) {
        sorted_weights[i] =  weights[i];
    }

    std::sort(sorted_weights.begin(), sorted_weights.end(), std::greater<T>());
    T max_bound = sorted_weights[0];
    T min_bound = sorted_weights[total_weights - 1];
    int max_index = 0;
    int min_index = total_weights - 1;


    // Loop thought the sorted weights until we reach the count value
    for (int i = 0; i < max_count; i++) {
        // For each iteration, move to the next element starting from the ends
        // of the sorted weights array
        max_index++;
        min_index--;
        max_bound = sorted_weights[max_index];
        min_bound = sorted_weights[min_index];
    }

    // Check which bound is closer to the zero value
    T limit = (fabs(min_bound) < fabs(max_bound)) ? max_bound : min_bound;
    int limit_index = (fabs(min_bound) < fabs(max_bound)) ? max_index : total_weights - min_index - 1;
    limit = fabs(limit);
    std::cout << "Limit value: " << limit << std::endl;
    std::cout << "Cutout percentage: " << (float)(total_weights - 2*limit_index)/(float)total_weights << std::endl;

    // Set the resolution value, so that the limit value is included in the FSR
    T levels = (T)wqpar.levels;
    return (T) (2/(levels-1))*(limit/bound);
}



template <typename T>
__global__ void WeightQuantizerCuda<T>::apply(T *weights, const WeightQuantizerParameter<T> &wqpar) {
  
    // int nthreads = context_->getNThreads();
    // int nblocks = context_->getNBlocks(size_, nthreads);
    auto s = context_->getStream();

    // For now, only the implementation for the uniform quantization is provided (no stochastic rounding)
    switch (wqpar.quantizer_type) {
        case WeightQuantizerType::Uniform: {
            if (wqpar.resolution > 0){
                // First, rescale the weights based on the maximum absolute value:
                // 1. Find the maximum absolute value of the weights
                if (amaximizer_ == nullptr){
                    amaximizer_ = RPU::make_unique<Maximizer<T>>(context_, size_);
                }
                amaximizer_->compute(weights, 1, true);
                T bound_value_;
                amaximizer_->copyMaxValuesToHost(&bound_value_);
                // 2. Rescale the weights
                RPU::math::elemscale(context_, weights, size_, (T)1.0 / bound_value_);

                (T) resolution = fit(weights, wqpar, bound_value_);
                // Quantize the weights
                RPU::math::uquantize(context_, weights, size_, resolution, wqpar.levels);

                // Rescale back the weights
                RPU::math::elemscale(context_, weights, size_, bound_value_);

            }
            break;
        }
        default:
            RPU_FATAL("Weight quantizer type not implemented.");
    }
}

template class WeightQuantizerCuda<float>;
#ifdef RPU_USE_DOUBLE
template class WeightQuantizerCuda<double>;
#endif
#ifdef RPU_USE_HALF
template class WeightQuantizerCuda<half>;
#endif

} // namespace RPU